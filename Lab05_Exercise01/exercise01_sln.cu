#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define N 4194304
#define THREADS_PER_BLOCK 128
#define PUMP_RATE 2

#define READ_BYTES N*(2*4)  //2 reads of 4 bytes (a and b)
#define WRITE_BYTES N*(4*1) //1 write of 4 bytes (to c)

__device__ float d_a[N];
__device__ float d_b[N];
__device__ float d_c[N];

void random_floats(float *a);

__global__ void vectorAdd() {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	d_c[i] = d_a[i] + d_b[i];
}

int main(void) {
	float *a, *b, *c;		// host copies of a, b, c
	int size = N * sizeof(float);
	hipEvent_t start, stop;
	float milliseconds = 0;
	int deviceCount = 0;
	double theoretical_BW;
	double measure_BW;


	hipGetDeviceCount(&deviceCount);
	if (deviceCount > 0)
	{
		hipSetDevice(0);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, 0);
		theoretical_BW = deviceProp.memoryClockRate * PUMP_RATE * (deviceProp.memoryBusWidth / 8.0) / 1e6; //convert to GB/s
	}


	hipEventCreate(&start);
	hipEventCreate(&stop);

	a = (float *)malloc(size); random_floats(a);
	b = (float *)malloc(size); random_floats(b);
	c = (float *)malloc(size);

	hipMemcpyToSymbol(HIP_SYMBOL(d_a), a, size);
	hipMemcpyToSymbol(HIP_SYMBOL(d_b), b, size);

	hipEventRecord(start);
	vectorAdd << <N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&milliseconds, start, stop);
	measure_BW = (READ_BYTES + WRITE_BYTES) / (milliseconds * 1e6);

	hipMemcpyFromSymbol(c, HIP_SYMBOL(d_c), size);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	free(a); free(b); free(c);

	printf("Execution time is %f ms\n", milliseconds);
	printf("Theoretical Bandwidth is %f GB/s\n", theoretical_BW);
	printf("Measured Bandwidth is %f GB/s\n", measure_BW);
	return 0;
}

void random_floats(float *a)
{
	for (unsigned int i = 0; i < N; i++){
		a[i] = (float)rand()/RAND_MAX;
	}
}
